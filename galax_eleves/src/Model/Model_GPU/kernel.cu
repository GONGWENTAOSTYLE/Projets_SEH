
#include <hip/hip_runtime.h>
#ifdef GALAX_MODEL_GPU

#include "cuda.h"
#include "kernel.cuh"
#include <mipp.h>
#define DIFF_T (0.1f)
#define EPS (1.0f)

inline __host__ __device__ float3 sub(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
inline __host__ __device__ float3 add(float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
 
inline __host__ __device__ float3 multi1(float3 a, float3 b)
{
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}

inline __host__ __device__ float3 multi2(float3 a, float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__global__ void compute_acc(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU, float* massesGPU, int n_particles)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	for (int j = 0; j < n_particles; j++)
	{
		if(i != j)
		{
			const float3 diff = sub(positionsGPU[j] , positionsGPU[i]);

			float3 dij3 = multi1(diff,diff);
			float dij = dij3.x + dij3.y + dij3.z;
			
			if (dij < 1.0)
			{
				dij = 10.0;
			}
			else
			{
				dij = std::sqrt(dij);
				dij = 10.0 / (dij * dij * dij);
			}
			float3 n = multi2(diff, dij);
			float3 m = multi2(n,massesGPU[j]);
			accelerationsGPU[i] = add(accelerationsGPU[i],m);
			
		}
	}
	/*
	const mipp::Reg<float> rpos_i = &positionsGPU[i];
    mipp::Reg<float> racc_i = &accelerationsGPU[i];
	const mipp::Reg<float> value =1.0;
    const mipp::Reg<float> zero =0.0;
    const mipp::Reg<float> G =10.0;	
    using T = float;
	constexpr int N = mipp::N<T>();
    auto vecLoopSize = (n_particles / N)* N;
	for (int j = 0; j < vecLoopSize; j += N)
	{
		const mipp::Reg<float3> rpos_j = &positionsGPU[j];
		mipp::Reg<float> masses_j = &massesGPU[j];
		if(i != j)
		{
 
			const mipp::Reg<float3>  diff = sub(rpos_j, rpos_i);
			const mipp::Reg<float3> dij3 = multi1(diff,diff);
			const mipp::Reg<float> dij = dij3.x + dij3.y + dij3.z;
			mipp::Msk<N> msk = (dij < value);
			dij = mipp::mask<float, mipp::mul>(msk, dij, dij, zero);
			dij = mipp::mask<float, mipp::add>(msk, dij, dij, value);
			dij = mipp::sqrt(dij);

			dij = G / (dij * dij * dij);
			
			float3 n = multi2(diff, dij);
			float3 m = multi2(n,masses_j);
			racc_i= add(racc_i,m);
			
		}
	}
	racc_i.store(&accelerationsGPU[i]);
	*/
}

__global__ void maj_pos(float3 * positionsGPU, float3 * velocitiesGPU, float3 * accelerationsGPU)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	velocitiesGPU[i] = add(velocitiesGPU[i], multi2(accelerationsGPU[i], 2.0f));
	positionsGPU[i] = add(positionsGPU[i], multi2(velocitiesGPU[i], 0.1f));

}

void update_position_cu(float3* positionsGPU, float3* velocitiesGPU, float3* accelerationsGPU, float* massesGPU, int n_particles)
{
	int nthreads = 128;
	int nblocks =  (n_particles + (nthreads -1)) / nthreads;

	compute_acc<<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU, massesGPU, n_particles);
	maj_pos    <<<nblocks, nthreads>>>(positionsGPU, velocitiesGPU, accelerationsGPU);
}


#endif // GALAX_MODEL_GPU